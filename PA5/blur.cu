#include "hip/hip_runtime.h"
/*
 * blur.cu
 * ------
 *
 * Definitions for the blurPixel and blurImage functions.
 *
 * Written by:
 *   Emmanuel Thiessen (thiessen@mcmaster.ca, MSID#000945053)
 *   James Lee            (leejr@mcmaster.ca, MSID#001318125)
 *
 * for SFWR ENG 4F03 PA5 (Winter 2017)
 */

#include "blur.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void blurOnePixel(Image *srcImage, Image *dstImage, int rad)
{
	/*
	int myIDx = blockIdx.x;
	int myIDy = blockIdx.y;
	printf("Hello world from x:%i, y:%i\n", myIDx, myIDy);
	*/

	int x = blockIdx.x;
	int y = blockIdx.y;
	unsigned long int
		pixel[3] = {0, 0, 0};
	int
		i, j, k,
		xmin = x - rad,
		xmax = x + rad,
		ymin = y - rad,
		ymax = y + rad,
		blurAreaSize;

	if (xmin < 0) xmin = 0;
	if (ymin < 0) ymin = 0;
	if (xmax >= srcImage->width)  xmax = srcImage->width  - 1;
	if (ymax >= srcImage->height) ymax = srcImage->height - 1;

	blurAreaSize = (xmax - xmin + 1) * (ymax - ymin + 1);

	for (j = ymin; j <= ymax; j++)
	for (i = xmin; i <= xmax; i++)
	for (k = 0; k < 3; k++)
	{
		pixel[k] += GetPixel(srcImage, i, j, k);
	}

	for (k = 0; k < 3; k++)
		SetPixel(dstImage, x, y, k, (unsigned char)(pixel[k] / blurAreaSize));
}


__host__ __device__ void
SetPixel(Image *image, int x, int y, int chan, unsigned char val)
{
	int offset = (y * image->width + x) * 3 + chan;

	image->data[offset] = val;
}


__host__ __device__ unsigned  char
GetPixel(Image *image, int x, int y, int chan)
{
	int offset = (y * image->width + x) * 3 + chan;

	return image->data[offset];
}


void blurPixel(Image *srcImage, Image *dstImage, int rad, int x, int y)
{
	unsigned long int
		pixel[3] = {0, 0, 0};
	int
		i, j, k,
		xmin = x - rad,
		xmax = x + rad,
		ymin = y - rad,
		ymax = y + rad,
		blurAreaSize;

	if (xmin < 0) xmin = 0;
	if (ymin < 0) ymin = 0;
	if (xmax >= srcImage->width)  xmax = srcImage->width  - 1;
	if (ymax >= srcImage->height) ymax = srcImage->height - 1;

	blurAreaSize = (xmax - xmin + 1) * (ymax - ymin + 1);

	for (j = ymin; j <= ymax; j++)
	for (i = xmin; i <= xmax; i++)
	for (k = 0; k < 3; k++)
	{
		pixel[k] += ImageGetPixel(srcImage, i, j, k);
	}

	for (k = 0; k < 3; k++)
		ImageSetPixel(dstImage, x, y, k, (unsigned char)(pixel[k] / blurAreaSize));
}

void blurImage(Image *srcImage, Image *dstImage, int rad)
{
	/*
	int i, j;

	for (j = 0; j < srcImage->height; j++)
	for (i = 0; i < srcImage->width;  i++)
	{
		blurPixel(srcImage, dstImage, rad, i, j);
	}
	*/

	dim3 block(1, 1);
	dim3 grid(srcImage->height, srcImage->width);
	blurOnePixel<<<grid, block>>>(srcImage, dstImage, rad);
	hipDeviceSynchronize();
}
