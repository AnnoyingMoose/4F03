#include "hip/hip_runtime.h"
/*
 * blur.cu
 * -------
 *
 * Definitions for the blurPixel and blurImage functions.
 *
 * Written by:
 *   Emmanuel Thiessen (thiessen@mcmaster.ca, MSID#000945053)
 *   James Lee            (leejr@mcmaster.ca, MSID#001318125)
 *
 * for SFWR ENG 4F03 PA5 (Winter 2017)
 */

#include "blur.h"

#include <assert.h>
#include <hip/hip_runtime.h>

__device__ void blurPixel(int rad, int width, int height, char *src, char *dst, int x, int y)
{
	unsigned long int
		pixel[3] = {0, 0, 0};
	int
		i, j, k,
		xmin = x - rad,
		xmax = x + rad,
		ymin = y - rad,
		ymax = y + rad,
		blurAreaSize;

	if (xmin < 0) xmin = 0;
	if (ymin < 0) ymin = 0;
	if (xmax >= width)  xmax = width  - 1;
	if (ymax >= height) ymax = height - 1;

	blurAreaSize = (xmax - xmin + 1) * (ymax - ymin + 1);

	for (j = ymin; j <= ymax; j++)
	for (i = xmin; i <= xmax; i++)
	for (k = 0; k < 3; k++)
	{
		//~ pixel[k] += ImageGetPixel(srcImage, i, j, k);
		pixel[k] += src[j * width * 3 + i * 3 + k];
	}

	for (k = 0; k < 3; k++)
		//~ ImageSetPixel(dstImage, x, y, k, (unsigned char)(pixel[k] / blurAreaSize));
		dst[j * width * 3 + i * 3 + k] = (unsigned char)(pixel[k] / blurAreaSize);
}

__global__ void gpuBlurImage(int rad, int width, int height, char *src, char *dst)
{
	int i, j, xmin, xmax, ymin, ymax;

	// Flatten grid/block co-ordinates
	tid =

	for (j = 0; j < height; j++)
	for (i = 0; i < width;  i++)
	{
		blurPixel(rad, width, height, src, dst, i, j);
	}
}

void blurImage(Image *srcImage, Image *dstImage, int rad)
{
	int
		width  = srcImage->width,
		height = srcImage->height;

	assert(width == dstImage->width && height == dstImage->height);

	dim3 b(32, 32, 1);
	dim3 g(ceil(width / 32.0), ceil(height / 32.0), 1);

	size_t sz = width * height * 3;
	char *src, *dst;

	hipMalloc(&src, sz);
	hipMalloc(&dst, sz);
	hipMemcpy(src, srcImage->data, sz, hipMemcpyHostToDevice);

	gpuBlurImage<<<g, b>>>(rad, width, height, src, dst);
	hipDeviceSynchronize();

	hipMemcpy(dstImage->data, dst, sz, hipMemcpyDeviceToHost);
	hipFree(src);
	hipFree(dst);
}
