#include "hip/hip_runtime.h"
/*
 * blur.cu
 * -------
 *
 * Definitions for the blurPixel and blurImage functions.
 *
 * Written by:
 *   Emmanuel Thiessen (thiessen@mcmaster.ca, MSID#000945053)
 *   James Lee            (leejr@mcmaster.ca, MSID#001318125)
 *
 * for SFWR ENG 4F03 PA5 (Winter 2017)
 */

extern "C"
{
#include "blur.h"
}

#include <assert.h>
#include <hip/hip_runtime.h>

__device__ void blurPixel(int rad, int width, int height, char *src, char *dst, int x, int y)
{
	int
		i, j,
		xmin = x - rad,
		xmax = x + rad,
		ymin = y - rad,
		ymax = y + rad,
		blurAreaSize;

	double red = 0;
	double green = 0;
	double blue = 0;
	int fields_count = 0;

	if (xmin < 0) xmin = 0;
	if (ymin < 0) ymin = 0;
	if (xmax >= width)  xmax = width  - 1;
	if (ymax >= height) ymax = height - 1;

	blurAreaSize = (xmax - xmin + 1) * (ymax - ymin + 1);

	for (j = ymin; j <= ymax; j++)
	for (i = xmin; i <= xmax; i++)
	{
		int currentOffset = (x + i + j * blurAreaSize)*3;
		red += src[currentOffset];
		green += src[currentOffset+1];
		blue += src[currentOffset+2];
		fields_count++;
	}

	dst[x*3] = red/fields_count;
	dst[x*3+1] = green/fields_count;
	dst[x*3+2] = blue/fields_count;
}

__global__ void gpuBlurImage(int rad, int width, int height, char *src, char *dst)
{
	int
		x = blockIdx.x * blockDim.x + threadIdx.x,
		y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
		blurPixel(rad, width, height, src, dst, x, y);
}

extern "C"
{
	void blurImage(Image *srcImage, Image *dstImage, int rad)
	{
		int
			width  = srcImage->width,
			height = srcImage->height;

		assert(width == dstImage->width && height == dstImage->height);

		dim3 b(1, 1);
		dim3 g(width, height);

		size_t sz = width * height * 3;
		char *src, *dst;

		hipMalloc(&src, sz);
		hipMalloc(&dst, sz);
		hipMemcpy(src, srcImage->data, sz, hipMemcpyHostToDevice);

		gpuBlurImage<<<g, b>>>(rad, width, height, src, dst);
		hipDeviceSynchronize();

		hipMemcpy(dstImage->data, dst, sz, hipMemcpyDeviceToHost);
		hipFree(src);
		hipFree(dst);
	}
}
