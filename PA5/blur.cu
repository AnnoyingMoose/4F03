#include "hip/hip_runtime.h"
/*
 * blur.cu
 * -------
 *
 * Definitions for the blurPixel and blurImage functions.
 *
 * Written by:
 *   Emmanuel Thiessen (thiessen@mcmaster.ca, MSID#000945053)
 *   James Lee            (leejr@mcmaster.ca, MSID#001318125)
 *
 * for SFWR ENG 4F03 PA5 (Winter 2017)
 */

extern "C"
{
#include "blur.h"
}

#include <assert.h>
#include <hip/hip_runtime.h>

__device__ void blurPixel(int rad, int width, int height, char *src, char *dst, int x, int y)
{
	unsigned long int
		pixel[3] = {0, 0, 0};
	int
		i, j, k,
		xmin = x - rad,
		xmax = x + rad,
		ymin = y - rad,
		ymax = y + rad,
		blurAreaSize;

	if (xmin < 0) xmin = 0;
	if (ymin < 0) ymin = 0;
	if (xmax >= width)  xmax = width  - 1;
	if (ymax >= height) ymax = height - 1;

	blurAreaSize = (xmax - xmin + 1) * (ymax - ymin + 1);

	for (j = ymin; j <= ymax; j++)
	for (i = xmin; i <= xmax; i++)
	for (k = 0; k < 3; k++)
	{
		//~ pixel[k] += ImageGetPixel(srcImage, i, j, k);
		pixel[k] += src[j * width * 3 + i * 3 + k];
	}

	for (k = 0; k < 3; k++)
		//~ ImageSetPixel(dstImage, x, y, k, (unsigned char)(pixel[k] / blurAreaSize));
		dst[y * width * 3 + x * 3 + k] = (unsigned char)(pixel[k] / blurAreaSize);
}

__global__ void gpuBlurImage(int rad, int width, int height, char *src, char *dst)
{
	int
		x = blockIdx.x * blockDim.x + threadIdx.x,
		y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height)
		blurPixel(rad, width, height, src, dst, x, y);
}

extern "C"
{
	void blurImage(Image *srcImage, Image *dstImage, int rad)
	{
		int
			width  = srcImage->width,
			height = srcImage->height;

		assert(width == dstImage->width && height == dstImage->height);

		dim3 b(32, 32, 1);
		dim3 g(ceil(width / 32.0), ceil(height / 32.0), 1);

		size_t sz = width * height * 3;
		char *src, *dst;

		hipMalloc(&src, sz);
		hipMalloc(&dst, sz);
		hipMemcpy(src, srcImage->data, sz, hipMemcpyHostToDevice);

		gpuBlurImage<<<g, b>>>(rad, width, height, src, dst);
		hipDeviceSynchronize();

		hipMemcpy(dstImage->data, dst, sz, hipMemcpyDeviceToHost);
		hipFree(src);
		hipFree(dst);
	}
}
